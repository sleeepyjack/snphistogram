#include <iostream>
#include <fstream>
#include "include/snpreader/SNPReader.h"
#include "include/batchreduce/include/batch_reduce.cuh"
#include "include/batchreduce/include/reduce_functors.cuh"
#include "include/cxxopts/src/cxxopts.hpp"

template <
  typename index_t,
  typename array_t
  >
void convert_data(SNPReader* reader, array_t* cases, index_t sizeCases, array_t* ctrls, index_t sizeCtrls)
{
  vector<SNP*> snps = reader->getSnpSet();

  for(index_t i=0; i < reader->getNumSnp(); i++)
  {
    memcpy(cases+(i*3*sizeCases)+(0*sizeCases), snps[i]->_case0Values, sizeof(array_t)*sizeCases);
    memcpy(cases+(i*3*sizeCases)+(1*sizeCases), snps[i]->_case1Values, sizeof(array_t)*sizeCases);
    memcpy(cases+(i*3*sizeCases)+(2*sizeCases), snps[i]->_case2Values, sizeof(array_t)*sizeCases);

    memcpy(ctrls+(i*3*sizeCtrls)+(0*sizeCtrls), snps[i]->_ctrl0Values, sizeof(array_t)*sizeCtrls);
    memcpy(ctrls+(i*3*sizeCtrls)+(1*sizeCtrls), snps[i]->_ctrl1Values, sizeof(array_t)*sizeCtrls);
    memcpy(ctrls+(i*3*sizeCtrls)+(2*sizeCtrls), snps[i]->_ctrl2Values, sizeof(array_t)*sizeCtrls);
  }
}

int main(int argc, char * argv[]) {
    typedef uint64_t index_t;
    typedef uint32_t array_t;
    // argparser
    cxxopts::Options options(argv[0], "Generate genotype histograms from SNPs");
    options.add_options()
        ("p,tped", "TPED file", cxxopts::value<std::string>(), "FILE")
        ("f,tfam", "TFAM file", cxxopts::value<std::string>(), "FILE")
        ("v,verbose", "Verbosity")
        ("o,out", "Output file", cxxopts::value<std::string>()->default_value("histograms.out")->implicit_value("histograms.out"));
    options.parse(argc, argv);

    // read input files
    SNPReader * reader = new SNPReader(options["tped"].as<std::string>().c_str(), options["tfam"].as<std::string>().c_str());
    reader->loadSNPSet();

    const index_t n = reader->getNumSnp();
    const index_t sizeCases = (reader->getNumCases()+32-1)/32;
    const index_t sizeCtrls = (reader->getNumCtrls()+32-1)/32;

    array_t * cases_h = (array_t*)malloc(sizeof(array_t)*3*sizeCases*n);
    array_t * ctrls_h = (array_t*)malloc(sizeof(array_t)*3*sizeCtrls*n);
    array_t * cases_d; hipMalloc(&cases_d, sizeof(array_t)*3*sizeCases*n);
    array_t * ctrls_d; hipMalloc(&ctrls_d, sizeof(array_t)*3*sizeCtrls*n);

    convert_data(reader, cases_h, sizeCases, ctrls_h, sizeCtrls);

    hipMemcpy(cases_d, cases_h, sizeof(array_t)*3*sizeCases*n, hipMemcpyHostToDevice);
    hipMemcpy(ctrls_d, ctrls_h, sizeof(array_t)*3*sizeCtrls*n, hipMemcpyHostToDevice);

    typedef sum_op_t<array_t> op_t;
    typedef BatchReduce<index_t, array_t, op_t> reduce_t;
    reduce_t reduce = reduce_t();

    array_t * histCases_h = (array_t*)malloc(sizeof(array_t)*3*n);
    array_t * histCtrls_h = (array_t*)malloc(sizeof(array_t)*3*n);
    array_t * histCases_d; hipMalloc(&histCases_d, sizeof(array_t)*3*n);
    array_t * histCtrls_d; hipMalloc(&histCtrls_d, sizeof(array_t)*3*n);

    // generate histograms
    reduce(cases_d, sizeCases, 3*n, histCases_d);
    reduce(ctrls_d, sizeCtrls, 3*n, histCtrls_d);

    hipMemcpy(histCases_h, histCases_d, sizeof(array_t)*3*n, hipMemcpyDeviceToHost);
    hipMemcpy(histCtrls_h, histCtrls_d, sizeof(array_t)*3*n, hipMemcpyDeviceToHost);

    // write histograms to file
    ofstream myfile;
    myfile.open(options["out"].as<std::string>());
    for (index_t i = 0; i < 3*n; i+=3)
    {
        myfile << histCases_h[i] << "\t" << histCases_h[i+1] << "\t" << histCases_h[i+2] << "\t" << histCtrls_h[i] << "\t" << histCtrls_h[i+1] << "\t" << histCtrls_h[i+2] << "\n";
    }
    if(options.count("v"))
    {
        cout << "\ncases\t\t\t|\tctrls\n";
        for (index_t i = 0; i < 3*n; i+=3)
        {
            cout   << histCases_h[i] << "\t" << histCases_h[i+1] << "\t" << histCases_h[i+2] << "\t|\t" << histCtrls_h[i] << "\t" << histCtrls_h[i+1] << "\t" << histCtrls_h[i+2] << "\n";
        }
    }
    myfile.close();

    // free memory
    free(cases_h);
    free(ctrls_h);
    free(histCases_h);
    free(histCtrls_h);
    hipFree(cases_d);
    hipFree(ctrls_d);
    hipFree(histCases_d);
    hipFree(histCtrls_d);
    delete reader;
}
